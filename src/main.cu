#include <cassert>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>


using namespace std;

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
  if (hipSuccess != err)
  {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
            file, line, (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__global__ void vecAdd(float *a, float *b, float *c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

int main()
{
  const int n_blocks = 128;
  const int n_threads = 512;
  const int n = n_blocks * n_threads;

  vector<float> h_a(n);
  vector<float> h_b(n);
  vector<float> h_c(n);

  for (int i = 0; i < n; ++i)
  {
    h_a[i] = i;
    h_b[i] = 2 * i;
    h_c[i] = 0;
  }

  float *d_a = nullptr;
  float *d_b = nullptr;
  float *d_c = nullptr;

  size_t array_size = sizeof(float) * n;

  checkCudaErrors(hipMalloc(&d_a, array_size));
  checkCudaErrors(hipMalloc(&d_b, array_size));
  checkCudaErrors(hipMalloc(&d_c, array_size));

  checkCudaErrors(hipMemcpy(d_a, h_a.data(), array_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_b, h_b.data(), array_size, hipMemcpyDefault));

  vecAdd<<<n_blocks, n_threads>>>(d_a, d_b, d_c);

  checkCudaErrors(hipMemcpy(h_c.data(), d_c, array_size, hipMemcpyDefault));

  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_b));
  checkCudaErrors(hipFree(d_c));
  d_a = nullptr;
  d_b = nullptr;
  d_c = nullptr;

  for (int i = 0; i < n; ++i)
  {
    assert(h_c[i] == (h_a[i] + h_b[i]));
  }
  cout << "OK!" << endl;

  return 0;
}